#include "hip/hip_runtime.h"
#include "SolveHeatEq.h"



SolveHeatEq::~SolveHeatEq()
{
    //dtor
}

void SolveHeatEq::init(const char *filename)
{
    tinyxml2::XMLDocument xmlDoc;
    tinyxml2::XMLError eResult = xmlDoc.LoadFile(filename);
    if (eResult != tinyxml2::XML_SUCCESS)
    {
        printf("XMLERROR is %d\nXML loading unsuccessfull.\n", eResult);
        exit(eResult);
    }

    /* Температура для всех cell */
    tinyxml2::XMLElement* pNode1 = xmlDoc.FirstChildElement("regions");
    if (pNode1 == nullptr)
    {
        printf("XMLERROR: No such FirstChildElement.\n");
        exit(1);
    }
    tinyxml2::XMLElement* pNode2 = pNode1->FirstChildElement("region");
    if (pNode2 == nullptr)
    {
        printf("XMLERROR: No such FirstChildElement.\n");
        exit(1);
    }
    tinyxml2::XMLElement* pNode3 = pNode2->FirstChildElement("parameters");
    if (pNode3 == nullptr)
    {
        printf("XMLERROR: No such FirstChildElement.\n");
        exit(1);
    }
    tinyxml2::XMLElement* pNode4 = pNode3->FirstChildElement("T");
    if (pNode4 == nullptr)
    {
        printf("XMLERROR: No such FirstChildElement.\n");
        exit(1);
    }
    double t;
    double k;

    eResult = pNode4->QueryDoubleAttribute("value", &t);
    if (eResult != tinyxml2::XML_SUCCESS)
        {
            printf("XMLERROR is %d\nXML loading unsuccessfull.\n", eResult);
            exit(eResult);
        }

    tinyxml2::XMLElement* pNode5 = pNode3->FirstChildElement("k");
    eResult = pNode5->QueryDoubleAttribute("value", &k);
    if (eResult != tinyxml2::XML_SUCCESS)
    {
            printf("XMLERROR is %d\nXML loading unsuccessfull.\n", eResult);
            exit(eResult);
    }
    for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
    {
        it->cellT.T = t;
        it->cellT.k = k;
    }
    /* Температура для заданных face */
    tinyxml2::XMLElement* pBnd = xmlDoc.FirstChildElement("boundaries");
    if (pBnd == nullptr)
    {
        printf("XMLERROR: No such FirstChildElement.\n");
        exit(1);
    }
    tinyxml2::XMLElement* pBndElement = pBnd->FirstChildElement("boundCond");
    if (pBndElement == nullptr)
    {
        printf("XMLERROR: No such FirstChildElement.\n");
        exit(1);
    }
    while (pBndElement != nullptr)
    {
        string str;
        tinyxml2::XMLElement* pName = pBndElement->FirstChildElement("name");
        if (pName == nullptr)
        {
            printf("XMLERROR: No such FirstChildElement.\n");
            exit(1);
        }
        str = pName->GetText();

        tinyxml2::XMLElement* pPar = pBndElement->FirstChildElement("parameters");
        if (pPar == nullptr)
        {
            printf("XMLERROR: No such FirstChildElement.\n");
            exit(1);
        }
        tinyxml2::XMLElement* pT = pPar->FirstChildElement("T");
        tinyxml2::XMLElement* pFlux = pPar->FirstChildElement("Flux");
        if (pT == nullptr && pFlux == nullptr)
        {
            printf("XMLERROR: No such FirstChildElement.\n");
            exit(1);
        }
        double temp;

        if(pT != nullptr)
        {
        	eResult = pT->QueryDoubleAttribute("value", &temp);
        	if (eResult != tinyxml2::XML_SUCCESS)
        	{
        		printf("XMLERROR is %d\nXML loading unsuccessfull.\n", eResult);
                exit(eResult);
            }

        	for (Mesh::FaceIterator it = msh->beginBndFace(str), ite = msh->endBndFace(str); it != ite; ++it)
        	{
        		it->bndT.T = temp;
        	}
        	bndNamesT.push_back(str);
        }
        else if(pFlux != nullptr)
        {
        	eResult = pFlux->QueryDoubleAttribute("value", &temp);
        	if (eResult != tinyxml2::XML_SUCCESS)
        	{
        		printf("XMLERROR is %d\nXML loading unsuccessfull.\n", eResult);
                exit(eResult);
            }

        	for (Mesh::FaceIterator it = msh->beginBndFace(str), ite = msh->endBndFace(str); it != ite; ++it)
        	{
        		it->bndT.Flux = temp;
        	}
        	bndNamesFlux.push_back(str);
        }

        pBndElement = pBndElement->NextSiblingElement("boundCond");
    }
}



__constant__ int const_sizeBndFlux;
__constant__ int const_sizeBndT;
__constant__ int const_sizeInnerFaces;
__constant__ int const_sizeCell;
__constant__ double const_k;
__constant__ double const_tau;


 __global__ void BndFacesFlux( double* cell_Flux, double* flux, int* indCell) {

	 int tid = threadIdx.x + blockIdx.x * blockDim.x;
	 	
	 while(tid < const_sizeBndFlux)
	 {
		 cell_Flux[ indCell[tid] ] += flux[tid];
		
		 tid += blockDim.x * gridDim.x;
	 }
}


 

 __global__ void BndFacesT( double* T, double* cell_Flux, double* h, double* S, double* bnd_T, int* indCell) {

	 int tid = threadIdx.x + blockIdx.x * blockDim.x;
	 
	 while(tid < const_sizeBndT)
	 {
		 cell_Flux[ indCell[tid] ] += S[tid] * (bnd_T[tid] - T[ indCell[tid] ]) / h[tid];
		 
		 tid += blockDim.x * gridDim.x;
	 }
 }


 

__global__ void InnerFaces( double* T, double* cell_Flux, double* h, double* S, int* indCell) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	double temp_value;

	while(tid < const_sizeInnerFaces ) {
		 temp_value = S[tid] * (T[ indCell[2 * tid + 1] ] - T[ indCell[2 * tid] ]) / h[tid];
		 cell_Flux[ indCell[2 * tid] ] += temp_value;
		 cell_Flux[ indCell[2 * tid + 1] ] -= temp_value;
		 tid += blockDim.x * gridDim.x;
	 }
}


__global__ void Cells( double* T, double* cell_Flux, double* V) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	while(tid < const_sizeCell) {
		 T[tid] += const_k * cell_Flux[tid] * const_tau / V[tid];
		 cell_Flux[tid] = 0;
		 tid += blockDim.x * gridDim.x;
	 }
}




void SolveHeatEq::calcHeatEquation(double t_max)
{
    double min_volume = msh->cells[0]->V;

    for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
    {
        double vol = it->V;
        if (min_volume > vol)
            min_volume = vol;
    }

    double k = msh->cells[0]->cellT.k;

    double tau = min_volume / (2.1 * k);

    double temp_value;
    double t = 0;

	// Cell'ы
    int sizeCell = msh->cells.size();

    double* T = (double*)malloc( sizeCell * sizeof(double) ); // Температура в Cell'е
    double* cell_Flux = (double*)malloc( sizeCell * sizeof(double) ); // Суммарный поток в Cell'е
    double* V = (double*)malloc( sizeCell * sizeof(double) ); // Объем Cell'а

    for(int i = 0; i < sizeCell; i++)
    {
    	T[i] = msh->cells[i]->cellT.T;
    	V[i] = msh->cells[i]->V;
    }

    memset(cell_Flux, 0, sizeCell * sizeof(double) );

    double* dev_T;
    hipMalloc( (void**)&dev_T, sizeCell * sizeof(double) );
    hipMemcpy( dev_T, T, sizeCell * sizeof(double), hipMemcpyHostToDevice );

    double* dev_V;
    hipMalloc( (void**)&dev_V, sizeCell * sizeof(double) );
    hipMemcpy( dev_V, V, sizeCell * sizeof(double), hipMemcpyHostToDevice );

    double* dev_cell_Flux;
    hipMalloc( (void**)&dev_cell_Flux, sizeCell * sizeof(double) );
    hipMemcpy( dev_cell_Flux, cell_Flux, sizeCell * sizeof(double), hipMemcpyHostToDevice );




	// внутренние face'ы
    int sizeInnerFaces = msh->inner_faces.size();

    double* inn_faces_h = (double*)malloc( sizeInnerFaces * sizeof(double) ); // расстояние между ячейками по данному Face'у
    double* inn_faces_S = (double*)malloc( sizeInnerFaces * sizeof(double) ); // Площадь
    int* inn_faces_indCell = (int*)malloc( 2 * sizeInnerFaces * sizeof(int) ); // inn_faces_indCell[ 2*i ], inn_faces_indCell[ 2*i + 1 ] - индексы смежных Cell'ов i-ого face'a в массивах T, cell_Flux, V
   
    for(int i = 0; i < sizeInnerFaces; i++)
    {
    	inn_faces_h[i] = msh->inner_faces[i]->h;
    	inn_faces_S[i] = msh->inner_faces[i]->S;
    	inn_faces_indCell[ 2*i ] = msh->inner_faces[i]->c[0]->index;
    	inn_faces_indCell[ 2*i + 1 ] = msh->inner_faces[i]->c[1]->index;
    }

    double* dev_inn_faces_h;
    hipMalloc( (void**)&dev_inn_faces_h,  sizeInnerFaces * sizeof(double));
    hipMemcpy( dev_inn_faces_h, inn_faces_h, sizeInnerFaces * sizeof(double), hipMemcpyHostToDevice );

    double* dev_inn_faces_S;
    hipMalloc( (void**)&dev_inn_faces_S,  sizeInnerFaces * sizeof(double));
    hipMemcpy( dev_inn_faces_S, inn_faces_S, sizeInnerFaces * sizeof(double), hipMemcpyHostToDevice );

    int* dev_inn_faces_indCell;
    hipMalloc( (void**)&dev_inn_faces_indCell, 2 * sizeInnerFaces * sizeof(int));
    hipMemcpy( dev_inn_faces_indCell, inn_faces_indCell, 2 * sizeInnerFaces * sizeof(int), hipMemcpyHostToDevice );

   

	// поток на границе
    int sizeBndFlux = 0; 
    for(vector<string>::iterator it = bndNamesFlux.begin(); it != bndNamesFlux.end(); ++it)
	{
		sizeBndFlux += msh->bnd_faces[(*it)].size();
	}


	double* flux_bnd_h = (double*)malloc( sizeBndFlux * sizeof(double) );
	double* flux_bnd_S = (double*)malloc( sizeBndFlux * sizeof(double) );
	double* bnd_flux = (double*)malloc( sizeBndFlux * sizeof(double) ); // поток
	int* flux_bnd_indCell = (int*)malloc( sizeBndFlux * sizeof(int) );  // flux_bnd_indCell[ i ] - индекс смежного Cell'a для i-ого face'a 

	int q = 0;
	for(vector<string>::iterator it = bndNamesFlux.begin(); it != bndNamesFlux.end(); ++it)
	{
		vector<Face*> vec_temp = msh->bnd_faces[(*it)];
		for(int i = 0; i < vec_temp.size(); i++)
		{
			flux_bnd_h[q] = vec_temp[i]->h;
			flux_bnd_S[q] = vec_temp[i]->S;
			bnd_flux[q] = vec_temp[i]->bndT.Flux;
			flux_bnd_indCell[q] = vec_temp[i]->c[0]->index;
			q++;
		}
	}

	double* dev_flux_bnd_h;
	hipMalloc( (void**)&dev_flux_bnd_h,  sizeBndFlux * sizeof(double));
	hipMemcpy( dev_flux_bnd_h, flux_bnd_h, sizeBndFlux * sizeof(double), hipMemcpyHostToDevice );

	double* dev_flux_bnd_S;
	hipMalloc( (void**)&dev_flux_bnd_S,  sizeBndFlux * sizeof(double));
	hipMemcpy( dev_flux_bnd_S, flux_bnd_S, sizeBndFlux * sizeof(double), hipMemcpyHostToDevice );

	double* dev_bnd_flux;
	hipMalloc( (void**)&dev_bnd_flux,  sizeBndFlux * sizeof(double));
	hipMemcpy( dev_bnd_flux, bnd_flux, sizeBndFlux * sizeof(double), hipMemcpyHostToDevice );

	int* dev_flux_bnd_indCell;
	hipMalloc( (void**)&dev_flux_bnd_indCell, sizeBndFlux * sizeof(int) );
	hipMemcpy( dev_flux_bnd_indCell, flux_bnd_indCell, sizeBndFlux * sizeof(int), hipMemcpyHostToDevice );



	// Температура на границе
	int sizeBndT = 0;
	for(vector<string>::iterator it = bndNamesT.begin(); it != bndNamesT.end(); ++it)
	{
		sizeBndT += msh->bnd_faces[(*it)].size();
	}


	double* T_bnd_h = (double*)malloc( sizeBndT * sizeof(double) ); // расстояние до фиктивной ячейки
	double* T_bnd_S = (double*)malloc( sizeBndT * sizeof(double) );
	double* bnd_T = (double*)malloc( sizeBndT * sizeof(double) ); // температура face'a
	int* T_bnd_indCell = (int*)malloc( sizeBndT * sizeof(int) ); // индекс Cell'a

	q = 0;
	for(vector<string>::iterator it = bndNamesT.begin(); it != bndNamesT.end(); ++it)
	{
		vector<Face*> vec_temp = msh->bnd_faces[(*it)];
		for(int i = 0; i < vec_temp.size(); i++)
		{
			T_bnd_h[q] = vec_temp[i]->h;
			T_bnd_S[q] = vec_temp[i]->S;
			bnd_T[q] = vec_temp[i]->bndT.T;
			T_bnd_indCell[q] = vec_temp[i]->c[0]->index;
			q++;
		}
	}

	double* dev_T_bnd_h;
	hipMalloc( (void**)&dev_T_bnd_h,  sizeBndT * sizeof(double) );
	hipMemcpy( dev_T_bnd_h, T_bnd_h, sizeBndT * sizeof(double), hipMemcpyHostToDevice );

	double* dev_T_bnd_S;
	hipMalloc( (void**)&dev_T_bnd_S,  sizeBndT * sizeof(double) );
	hipMemcpy( dev_T_bnd_S, T_bnd_S, sizeBndT * sizeof(double), hipMemcpyHostToDevice );

	double* dev_bnd_T;
	hipMalloc( (void**)&dev_bnd_T,  sizeBndT * sizeof(double) );
	hipMemcpy( dev_bnd_T, bnd_T, sizeBndT * sizeof(double), hipMemcpyHostToDevice );

	int* dev_T_bnd_indCell;
	hipMalloc( (void**)&dev_T_bnd_indCell, sizeBndT * sizeof(int));
	hipMemcpy( dev_T_bnd_indCell, T_bnd_indCell, sizeBndT * sizeof(int), hipMemcpyHostToDevice );

	
	hipMemcpyToSymbol(HIP_SYMBOL(const_k), &k, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(const_tau), &tau, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(const_sizeCell), &sizeCell, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(const_sizeInnerFaces), &sizeInnerFaces, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(const_sizeBndFlux), &sizeBndFlux, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(const_sizeBndT), &sizeBndT, sizeof(int), 0, hipMemcpyHostToDevice);
	

	int threads = 128;
	int blocks = 128;

    while (t < t_max)
    {	
        t += tau;
		
		// Поток на границе
		BndFacesFlux <<< blocks, threads >>>(dev_cell_Flux, dev_bnd_flux, dev_flux_bnd_indCell);
		// Поток на границе
		BndFacesT <<< blocks, threads >>>(dev_T, dev_cell_Flux, dev_T_bnd_h, dev_T_bnd_S, dev_bnd_T, dev_T_bnd_indCell);
		// Поток во внутренних face'ах
		InnerFaces <<< blocks, threads >>>(dev_T, dev_cell_Flux, dev_inn_faces_h, dev_inn_faces_S, dev_inn_faces_indCell);
		// Новое значение темрературы в ячейке
		Cells <<< blocks, threads >>>(dev_T, dev_cell_Flux, dev_V);
		
		/* Решение на CPU
        for (Mesh::BndFaceIterator it = msh->beginBndFace(&(msh->bnd_faces), &bndNamesFlux), ite = msh->endBndFace(&(msh->bnd_faces), &bndNamesFlux); it != ite; ++it)
        {
           	it->c[0]->cellT.Flux += it->bndT.Flux;
        }
		 
        
        for (Mesh::BndFaceIterator it = msh->beginBndFace(&(msh->bnd_faces), &bndNamesT), ite = msh->endBndFace(&(msh->bnd_faces), &bndNamesT); it != ite; ++it)
        {
        	it->c[0]->cellT.Flux += (it->S*(it->bndT.T - it->c[0]->cellT.T) / it->h);

        }
        
        for (Mesh::FaceIterator it = msh->beginInnerFace(), ite = msh->endInnerFace(); it != ite; ++it)
        {
        	temp_value = it->S*(it->c[1]->cellT.T - it->c[0]->cellT.T) / it->h;
        	it->c[0]->cellT.Flux += temp_value;
        	it->c[1]->cellT.Flux -= temp_value;

        }
		
        for (Mesh::CellIterator it = msh->beginCell(), ite = msh->endCell(); it != ite; ++it)
        {
        	it->cellT.T += it->cellT.Flux * tau * k / it->V;
        	it->cellT.Flux = 0;
        }
   		 */
    }

	hipMemcpy(T, dev_T, sizeCell * sizeof(double), hipMemcpyDeviceToHost);
	for (int i = 0; i < sizeCell; i++)
	{
		msh->cells[i]->cellT.T = T[i];
	}

}


void SolveHeatEq::save(const char *filename, const char *header)
{
    FILE *out;
    out = fopen(filename, "w");
    fprintf(out, "# vtk DataFile Version 3.0\n");
    //The header can be used to describe the data
    fprintf(out, "%s\n", header);
    fprintf(out, "ASCII\n");
    fprintf(out, "DATASET UNSTRUCTURED_GRID\n");
    fprintf(out, "POINTS %d double\n", msh->pCount);
    for (int i = 0; i < msh->pCount; i++)
    {
        fprintf(out, "%f %f %f\n", msh->points[i].x, msh->points[i].y, msh->points[i].z);
    }

    int cellCount = msh->cells.size();

    int cellSize = 0;//the size of the cell list (count of points in all cells)
    for (int i = 0; i < cellCount; i++)
    {
        cellSize += msh->cells[i]->pCount;
    }

    /*
    cellSize + cellCount :
    cellSize + one number for each cell - count of points in this cell
    */
    fprintf(out, "CELLS %d %d\n", cellCount, cellSize + cellCount);
    for (int i = 0; i < cellCount; i++)
    {
        fprintf(out, "%d", msh->cells[i]->pCount);
        for (int k = 0; k < msh->cells[i]->pCount; k++)
        {
            int ind = -1;
            Point* addr = msh->cells[i]->p[k];
            for (int j = 0; j < msh->pCount; j++)
            {
                if (&(msh->points[j]) == addr)
                    ind = j;
            }
            fprintf(out, " %d", ind);
        }
        fprintf(out, "\n");
    }
    fprintf(out, "CELL_TYPES %d\n", cellCount);
    for (int i = 0; i < cellCount; i++)
    {
        switch (msh->cells[i]->type)
        {
        case 111:
        {
            fprintf(out, "10\n"); //10 - VTK_TETRA
            break;
        }
        case 112:
        {
            fprintf(out, "13\n"); //13 - VTK_WEDGE
            break;
        }
        case 115:
        {
            fprintf(out, "12\n"); //12 - VTK_HEXAHEDRON
            break;
        }
        }

    }
    fprintf(out, "CELL_DATA %d\nSCALARS temperature double 1\nLOOKUP_TABLE default\n", cellCount);
    for (int i = 0; i < cellCount; i++)
    {
        fprintf(out, "%f\n", msh->cells[i]->cellT.T);
    }
    fclose(out);
}


